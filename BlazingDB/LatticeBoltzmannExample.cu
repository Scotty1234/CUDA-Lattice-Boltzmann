#include "LatticeBoltzmannExample.h"

#include "hip/hip_runtime.h"

__constant__ float relaxationDevice; //Probably not worth it for one float, but still, all threads will use this. 

void LatticeBoltzmannExample::cudaLBInit()
{
	//std::unique_ptr<ThrustVector<real>> thrustVectors(new ThrustVector<real>(simulationParameters->xLength, simulationParameters->yLength));

	thrustVectors = new ThrustVector<real>(simulationParameters->xLength, simulationParameters->yLength);

	hipMemcpyToSymbol(HIP_SYMBOL(&relaxationDevice), &(simulationParameters->relaxation), sizeof(simulationParameters->relaxation));
}

void LatticeBoltzmannExample::cudaRun()
{
	if (!isCudaCompatible())
	{
		exit(5);
	}



	cudaLBInit();
}

bool LatticeBoltzmannExample::isCudaCompatible()
{
	hipDeviceProp_t properties;

	int count;

	hipError_t hipError_t =hipGetDeviceCount(&count);
	
	if (hipError_t == hipErrorNoDevice)
	{
		std::cout << "Error: No cuda capable device found.\n";
		return false;
	}
	else if (hipError_t == hipErrorInsufficientDriver)
	{
		std::cout << "Error: Insufficient cuda driver.\n";
		return false;
	}

	return true;
}

void LatticeBoltzmannExample::waitForDevice()
{
	if (hipDeviceSynchronize() != hipSuccess)
		hipError_t();
}